#include "merkle_tree.hpp"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdgpu/memory.h>
#include <stdgpu/unordered_map.cuh>

void digest_to_hex_(const uint8_t digest[20], char* output) {
  int i,j;
  char* c = output;
  for(i=0; i<20/4; i++) {
    for(j=0; j<4; j++) {
      sprintf(c, "%02X", digest[i*4 + j]);
      c += 2;
    }
    sprintf(c, " ");
    c += 1;
  }
  *(c-1) = '\0';
}

void print_merkle_tree(uint8_t* tree, const size_t hash_len, const size_t num_leaves) {
  char buffer[80];
  int counter = 2;
  for(size_t i=0; i<2*num_leaves-1; i++) {
    digest_to_hex_(tree+i*hash_len, buffer);
    printf("Node: %zd: %s \n", i, buffer);
    if(i == counter) {
      printf("\n");
      counter += 2*counter;
    }
  }
}

int main(int argc, char** argv) {
  const char* test_str0 = "Hello Muddah. Hello Fadduh. Here I am at camp Granada"; //53
  const char* test_str1 = "Hello Mother. Hello Father. Here I am at camp Granada"; //53
//  const char* test_str0 = "aaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaa"; //53
  size_t data_len = 53;
  size_t chunk_size = 1;
  size_t num_leaves = data_len/chunk_size;
  if(chunk_size*num_leaves < data_len)
    num_leaves += 1;
  size_t num_nodes = 2*num_leaves-1;
  printf("Data length: %d\n", data_len);
  printf("Chunk size: %d\n", chunk_size);
  printf("Num leaves: %d\n", num_leaves);
  printf("Num nodes: %d\n", num_nodes);

  uint8_t* gpu_str0;
  hipMalloc(&gpu_str0, data_len);
  hipMemcpy(gpu_str0, test_str0, data_len, hipMemcpyHostToDevice);
  uint8_t* gpu_str1;
  hipMalloc(&gpu_str1, data_len);
  hipMemcpy(gpu_str1, test_str1, data_len, hipMemcpyHostToDevice);
  printf("Copied data to GPU\n");
  uint8_t* tree0 = (uint8_t*)malloc(num_nodes*20);
  uint8_t* tree0_d;
  hipMalloc(&tree0_d, num_nodes*20);
  uint8_t* tree1 = (uint8_t*)malloc(num_nodes*20);
  uint8_t* tree1_d;
  hipMalloc(&tree1_d, num_nodes*20);
  printf("Creating Merkle Tree\n");
  hipDeviceSynchronize();
  CreateMerkleTree(gpu_str0, data_len, chunk_size, tree0_d, GPU);
  CreateMerkleTree(gpu_str1, data_len, chunk_size, tree1_d, GPU);
//  CreateMerkleTree((uint8_t*)test_str0, 53, chunk_size, tree0, GPU);
//  CreateMerkleTree((uint8_t*)test_str1, 53, chunk_size, tree1, GPU);
  hipDeviceSynchronize();
  printf("Created Merkle Tree\n");
  hipMemcpy(tree0, tree0_d, num_nodes*20, hipMemcpyDeviceToHost);
  hipMemcpy(tree1, tree1_d, num_nodes*20, hipMemcpyDeviceToHost);
  printf("Copied data to CPU\n");
  
  print_merkle_tree(tree0, 20, num_leaves);
  print_merkle_tree(tree1, 20, num_leaves);
  
//  using DistinctMap = stdgpu::unordered_map<HashDigest, NodeInfo, transparent_sha1_hash>;
//  using SharedMap = stdgpu::unordered_map<uint32_t,uint32_t>;
//  DistinctMap distinct_map0 = DistinctMap::createDeviceObject(num_nodes);
//  DistinctMap distinct_map1 = DistinctMap::createDeviceObject(num_nodes);
//  SharedMap shared_map0 = SharedMap::createDeviceObject(num_nodes);
//  SharedMap shared_map1 = SharedMap::createDeviceObject(num_nodes);
//
//  FindDistinctSubtrees(tree0_d, num_nodes, 0, distinct_map0, shared_map0, GPU);
//
//  printf("Num distinct entries: %d\n", distinct_map0.size());

//  DistinctMap::destroyDeviceObject(distinct_map0);
//  DistinctMap::destroyDeviceObject(distinct_map1);
//  SharedMap::destroyDeviceObject(shared_map0);
//  SharedMap::destroyDeviceObject(shared_map1);
}
