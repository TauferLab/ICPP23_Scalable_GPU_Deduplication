#include "hip/hip_runtime.h"
#include "merkle_tree.hpp"
#include "helpers.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <string>
#include <cstring>
#include <sys/stat.h>
#include <stdgpu/memory.h>
#include <stdgpu/unordered_map.cuh>
#include <stdlib.h>
#include <chrono>

#define MERKLE

long get_file_size(std::string filename) {
  struct stat stat_buf;
  int rc = stat(filename.c_str(), &stat_buf);
  return rc == 0 ? stat_buf.st_size : -1;
}

void digest_to_hex_(const uint8_t digest[20], char* output) {
  int i,j;
  char* c = output;
  for(i=0; i<20/4; i++) {
    for(j=0; j<4; j++) {
      sprintf(c, "%02X", digest[i*4 + j]);
      c += 2;
    }
    sprintf(c, " ");
    c += 1;
  }
  *(c-1) = '\0';
}

void print_merkle_tree(uint8_t* tree, const unsigned int hash_len, const unsigned int num_leaves) {
  printf("============================================================\n");
  char buffer[80];
  unsigned int counter = 2;
  for(unsigned int i=0; i<2*num_leaves-1; i++) {
    digest_to_hex_(tree+i*hash_len, buffer);
    printf("Node: %u: %s \n", i, buffer);
    if(i == counter) {
      printf("\n");
      counter += 2*counter;
    }
  }
  printf("============================================================\n");
}

uint8_t* generate_data(const unsigned int len) {
  uint8_t* data = (uint8_t*)(malloc(len));
  srand(NULL);
  for(unsigned int i=0; i<len; i++) {
    data[i] = static_cast<uint8_t>(rand() % 256);
  }
  return data;
}

//uint8_t* copy_and_perturb(const uint8_t* data, const unsigned int len, const unsigned int chance_of_change) {
//  srand(NULL);
//  uint8_t* new_data = (uint8_t*)(malloc(len));
//  for(unsigned int i=0; i<len; i++) {
//    new_data[i] = data[i];
//    unsigned int roll = rand() % 100;
//    if(roll < chance_of_change)
//      new_data[i] = static_cast<uint8_t>(rand() % 128);
//  }
//  return new_data;
//}

enum PerturbMode {
  BlockStart=0,
  Random
};

uint8_t* copy_and_perturb(const uint8_t* data, const unsigned int len, const unsigned int chance_of_change, PerturbMode mode) {
  srand(time(NULL));
  unsigned int num_changes = static_cast<unsigned int>(len*(static_cast<float>(chance_of_change)/100.f));
  uint8_t* new_data = (uint8_t*)(malloc(len));
  std::memcpy(new_data, data, len);
  if(mode == BlockStart) {
    for(unsigned int i=0; i<num_changes; i++) {
      new_data[i] = static_cast<uint8_t>(rand() % 256);
    }
  } else if(mode == Random) {
    for(unsigned int i=0; i<num_changes; i++) {
      unsigned int pos = rand() % len;
      new_data[pos] = static_cast<uint8_t>(rand() % 256);
    }
  }
  return new_data;
}

__global__ void print_distinct_nodes(const stdgpu::unordered_map<HashDigest, NodeInfo, transparent_sha1_hash> distinct) {
  for(auto it=distinct.cbegin(); it!=distinct.cend(); it++) {
//    if(it->second != NodeInfo() && it->second != NodeInfo(0,0,0))
    if(distinct.contains(it->first))
    printf("(%u,%u,%u)\n", it->second.node, it->second.src, it->second.tree);
  }
}

int main(int argc, char** argv) {
  using Timer = std::chrono::high_resolution_clock;
//  if(argc < 3) {
//    printf("Expected more arguments.\n");
//    printf("./gpu_test chunk_size checkpoint_file\n");
//  }
//  int chunk_size = atoi(argv[1]);
//  std::string full_chkpt(argv[2]);
//  std::string incr_chkpt = full_chkpt + ".gpu_test.incr_chkpt";
//  std::vector<std::string> prev_chkpt;
//  for(int i=3; i<argc; i++) {
//    prev_chkpt.push_back(std::string(argv[i]));
//  }

  unsigned int chunk_size = static_cast<unsigned int>(strtoul(argv[1], NULL, 0));
  unsigned int data_len   = static_cast<unsigned int>(strtoul(argv[2], NULL, 0));
  unsigned int chance     = static_cast<unsigned int>(strtoul(argv[3], NULL, 0));

//  const char* test_str0 = "Hello Muddah. Hello Fadduh. Here I am at camp Granada"; //53
//  const char* test_str1 = "Hello Mother. Hello Father. Here I am at camp Granada"; //53
//  unsigned int data_len = 53;
//  unsigned int chunk_size = 1;

  unsigned int num_leaves = data_len/chunk_size;
  if(chunk_size*num_leaves < data_len)
    num_leaves += 1;
  unsigned int num_nodes = 2*num_leaves-1;
  printf("Data length: %u\n", data_len);
  printf("Chunk size: %u\n", chunk_size);
  printf("Num leaves: %u\n", num_leaves);
  printf("Num nodes: %u\n", num_nodes);

  uint8_t* test_str0 = generate_data(data_len);
  uint8_t* test_str1 = copy_and_perturb(test_str0, data_len, chance, BlockStart);

//printf("Test string 0: ");
//for(int i=0; i<data_len; i++) {
//  printf("%hhx", test_str0[i]);
//}
//printf("\n");
//printf("Test string 1: ");
//for(int i=0; i<data_len; i++) {
//  printf("%hhx", test_str1[i]);
//}
//printf("\n");

  uint8_t* gpu_str0;
  hipMalloc(&gpu_str0, data_len);
  hipMemcpy(gpu_str0, test_str0, data_len, hipMemcpyHostToDevice);
  uint8_t* gpu_str1;
  hipMalloc(&gpu_str1, data_len);
  hipMemcpy(gpu_str1, test_str1, data_len, hipMemcpyHostToDevice);
  printf("Copied data to GPU\n");
#ifdef MERKLE
  uint8_t* tree0 = (uint8_t*)malloc(num_nodes*20);
  uint8_t* tree0_d;
  hipMalloc(&tree0_d, num_nodes*20);
  uint8_t* tree1 = (uint8_t*)malloc(num_nodes*20);
  uint8_t* tree1_d;
  hipMalloc(&tree1_d, num_nodes*20);
  printf("Creating Merkle Tree\n");
  hipDeviceSynchronize();
#else
  uint32_t *hashlist0_d, *hashlist1_d;
  uint32_t* hashlist0_h = (uint32_t*)(malloc(num_leaves*20));
  uint32_t* hashlist1_h = (uint32_t*)(malloc(num_leaves*20));
  hipMalloc(&hashlist0_d, num_leaves*20);
  hipMalloc(&hashlist1_d, num_leaves*20);
  printf("Allocated hash lists\n");
#endif

for(int i=0; i<5; i++) {
#ifdef MERKLE
  Timer::time_point start_create_tree0 = Timer::now();
  CreateMerkleTree(gpu_str0, data_len, chunk_size, tree0_d, GPU);
  Timer::time_point end_create_tree0 = Timer::now();
  Timer::time_point start_create_tree1 = Timer::now();
  CreateMerkleTree(gpu_str1, data_len, chunk_size, tree1_d, GPU);
  Timer::time_point end_create_tree1 = Timer::now();
  hipDeviceSynchronize();
  printf("Created Merkle Tree\n");
  hipMemcpy(tree0, tree0_d, num_nodes*20, hipMemcpyDeviceToHost);
  hipMemcpy(tree1, tree1_d, num_nodes*20, hipMemcpyDeviceToHost);
  printf("Copied data to CPU\n");
#else
  Timer::time_point start_create_tree0 = Timer::now();
  CreateHashList(gpu_str0, data_len, hashlist0_d, chunk_size, num_leaves, GPU);
  Timer::time_point end_create_tree0 = Timer::now();
  Timer::time_point start_create_tree1 = Timer::now();
  CreateHashList(gpu_str1, data_len, hashlist1_d, chunk_size, num_leaves, GPU);
  Timer::time_point end_create_tree1 = Timer::now();
  hipDeviceSynchronize();
  printf("Created Hash list Tree\n");
  hipMemcpy(hashlist0_h, hashlist0_d, num_leaves*20, hipMemcpyDeviceToHost);
  hipMemcpy(hashlist1_h, hashlist1_d, num_leaves*20, hipMemcpyDeviceToHost);
  printf("Copied data to CPU\n");
#endif
  
////  print_merkle_tree(tree0, 20, num_leaves);
////  print_merkle_tree(tree1, 20, num_leaves);
//
//#ifdef MERKLE
//  HashTable<HashDigest, NodeInfo> distinct_map0(2*num_nodes);
//  HashTable<HashDigest, NodeInfo> distinct_map1(2*num_nodes);
//  printf("Pointer: %p\n", tree0_d);
//  FindDistinctSubtrees(tree0_d, num_nodes, 0, distinct_map0, GPU);
//  printf("Num distinct: %u\n", distinct_map0.size());
////  print_hash_table(distinct_map0.m_capacity_d, distinct_map0.m_available_indices_d);
//
//printf("Distinct map 0 capacity: %u\n", distinct_map0.capacity());
//printf("Distinct map 0 size: %u\n", distinct_map0.size());
//printf("Distinct map 1 capacity: %u\n", distinct_map1.capacity());
//printf("Distinct map 1 size: %u\n", distinct_map1.size());
//
//  printf("Pointer: %p\n", tree1_d);
//#ifdef MERKLE
//  FindDistinctSubtrees(tree1_d, num_nodes, 1, distinct_map1, GPU);
//#else
//#endif
//  printf("Num distinct: %u\n", distinct_map1.size());
////  print_hash_table(distinct_map1.m_capacity_d, distinct_map1.m_available_indices_d, distinct_map1.m_keys_d, distinct_map1.m_values_d);
//  
//printf("Distinct map 0 capacity: %u\n", distinct_map0.capacity());
//printf("Distinct map 0 size: %u\n", distinct_map0.size());
//printf("Distinct map 1 capacity: %u\n", distinct_map1.capacity());
//printf("Distinct map 1 size: %u\n", distinct_map1.size());
//  print_hash_table(distinct_map1.m_capacity_d, distinct_map1.m_available_indices_d, distinct_map1.m_keys_d, distinct_map1.m_values_d);
//#else
//  size_t *unique_chunks0, *unique_chunks1;
//  int *num_unique0, *num_unique1;
//  hipMalloc(&unique_chunks0, sizeof(size_t)*num_leaves);
//  hipMalloc(&num_unique0, sizeof(int)*num_leaves);
//  hipMalloc(&unique_chunks1, sizeof(size_t)*num_leaves);
//  hipMalloc(&num_unique1, sizeof(int)*num_leaves);
//  FindDistinctHashes(hashlist0_d, 20,  num_leaves, unique_chunks0, num_unique0, GPU);
//  FindDistinctHashes(hashlist1_d, 20,  num_leaves, unique_chunks1, num_unique1, GPU);
//#endif
//
//#ifdef MERKLE
//  CompareTrees(tree1_d, num_nodes, 1, distinct_map1, distinct_map0, GPU);
//printf("Distinct map 0 capacity: %u\n", distinct_map0.capacity());
//printf("Distinct map 0 size: %u\n", distinct_map0.size());
//printf("Distinct map 1 capacity: %u\n", distinct_map1.capacity());
//printf("Distinct map 1 size: %u\n", distinct_map1.size());
//
//  print_hash_table(distinct_map1.m_capacity_d, distinct_map1.m_available_indices_d, distinct_map1.m_keys_d, distinct_map1.m_values_d);
//printf("Done printing table\n");
//#else
//void ComparePriorHashes(const uint32_t* hashlist1_d,
//                        const size_t num_leaves,
//                        const uint32_t* hashlist0_d,
//                        const size_t num_leaves,
//                        const int 20, 
//                        const int num_unique_hashes,
//                        size_t* changed_regions,
//                        int* num_changes,
//                        Mode mode) {
//#endif
  

#ifdef MERKLE
printf("Allocating maps for Merkle trees\n");
  using DistinctMap = stdgpu::unordered_map<HashDigest, NodeInfo, transparent_sha1_hash>;
  using SharedMap = stdgpu::unordered_map<uint32_t,uint32_t>;
  DistinctMap distinct_map0 = DistinctMap::createDeviceObject(2*num_nodes);
  DistinctMap distinct_map1 = DistinctMap::createDeviceObject(2*num_nodes);
  SharedMap shared_map0 = SharedMap::createDeviceObject(num_nodes);
  SharedMap shared_map1 = SharedMap::createDeviceObject(num_nodes);
#else
printf("Allocating maps for hash lists\n");
  using DistinctMap = stdgpu::unordered_map<HashDigest, HashListInfo, transparent_sha1_hash>;
  using SharedMap = stdgpu::unordered_map<unsigned int, unsigned int>;
  DistinctMap distinct_map0 = DistinctMap::createDeviceObject(2*num_leaves);
  DistinctMap distinct_map1 = DistinctMap::createDeviceObject(2*num_leaves);
printf("Creating shared maps\n");
  SharedMap shared_map0 = SharedMap::createDeviceObject(num_nodes);
  SharedMap shared_map1 = SharedMap::createDeviceObject(num_nodes);
#endif

printf("Finding distinct hashes/nodes for string 0\n");
  Timer::time_point start_find_distinct0 = Timer::now();
#ifdef MERKLE
  FindDistinctSubtrees(tree0_d, num_nodes, 0, distinct_map0, shared_map0, GPU);
#else
  FindDistinctHashes(hashlist0_d, num_leaves, 0, distinct_map0, shared_map0, GPU);
#endif
  Timer::time_point end_find_distinct0 = Timer::now();

printf("Finding distinct hashes/nodes for string 1\n");
  Timer::time_point start_find_distinct1 = Timer::now();
#ifdef MERKLE
  FindDistinctSubtrees(tree1_d, num_nodes, 1, distinct_map1, shared_map1, GPU);
#else
  FindDistinctHashes(hashlist1_d, num_leaves, 1, distinct_map1, shared_map1, GPU);
#endif
  Timer::time_point end_find_distinct1 = Timer::now();

  printf("Num distinct entries (tree 0): %d\n", distinct_map0.size());
  printf("Num distinct entries (tree 1): %d\n", distinct_map1.size());
  printf("Num shared entries (tree 0): %d\n", shared_map0.size());
  printf("Num shared entries (tree 1): %d\n", shared_map1.size());

printf("Comparing trees/lists\n");
  Timer::time_point start_compare1 = Timer::now();
#ifdef MERKLE
  CompareTrees(tree1_d, num_nodes, 1, distinct_map1, distinct_map0, GPU);
#else
  ComparePriorHashes(hashlist1_d, num_leaves, distinct_map1, shared_map1, distinct_map0, GPU);
#endif
  Timer::time_point end_compare1 = Timer::now();

  printf("Num distinct entries (tree 0): %d\n", distinct_map0.size());
  printf("Num distinct entries (tree 1): %d\n", distinct_map1.size());
hipDeviceSynchronize();
//  print_distinct_nodes<<<1,1>>>(distinct_map1);

  printf("Timing info\n");
  printf("CreateTree, FindDistinctSubtrees, CompareTrees\n");
  std::cout << std::chrono::duration_cast<std::chrono::duration<double>>(end_create_tree0 - start_create_tree0).count();
  std::cout << ",";
  std::cout << std::chrono::duration_cast<std::chrono::duration<double>>(end_find_distinct0 - start_find_distinct0).count();
  std::cout << ",";
  std::cout << "N/A";

  std::cout << "\n";
  std::cout << std::chrono::duration_cast<std::chrono::duration<double>>(end_create_tree1 - start_create_tree1).count();
  std::cout << ",";
  std::cout << std::chrono::duration_cast<std::chrono::duration<double>>(end_find_distinct1 - start_find_distinct1).count();
  std::cout << ",";
  std::cout << std::chrono::duration_cast<std::chrono::duration<double>>(end_compare1 - start_compare1).count();
  std::cout << "\n";

  DistinctMap::destroyDeviceObject(distinct_map0);
  DistinctMap::destroyDeviceObject(distinct_map1);
  SharedMap::destroyDeviceObject(shared_map0);
  SharedMap::destroyDeviceObject(shared_map1);
}

}
